#include "hip/hip_runtime.h"
/*
    CUDA-Accelerated ACO algorithms for TSP and QAP
    Based on ACOTSPQAP by Manuel López-Ibáñez and Thomas Stützle
    
    This CUDA implementation parallelizes:
    1. Ant solution construction
    2. Pheromone matrix updates
    3. Distance/flow matrix computations
    4. Local search operations
*/

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_cooperative_groups.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <time.h>
#include <algorithm>

// Include the shared header
#include "cuda_aco.h"

// CUDA error checking macro
#define CUDA_CHECK(call) do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, \
                hipGetErrorString(error)); \
        exit(1); \
    } \
} while(0)

// Problem and algorithm parameters
#define MAX_CITIES 10000
#define MAX_ANTS 1024
#define BLOCK_SIZE 256
#define PHEROMONE_INIT 1.0f
#define ALPHA 1.0f  // Pheromone influence
#define BETA 2.0f   // Heuristic influence
#define RHO 0.5f    // Evaporation rate
#define Q0 0.9f     // ACS exploration/exploitation parameter

// Ant structure
typedef struct {
    int* tour;           // Solution tour
    bool* visited;       // Visited cities/locations
    float tour_length;   // Tour cost
    int current_city;    // Current position
    int tour_size;       // Number of cities in tour
} Ant;

// ACO data structure (internal implementation)
struct ACOData {
    // Problem data
    float* d_distance;      // Distance matrix (TSP) or flow matrix (QAP)
    float* d_flow;          // Second matrix for QAP
    float* d_pheromone;     // Pheromone matrix
    float* d_heuristic;     // Heuristic information matrix (1/distance for TSP)
    float* d_prob;          // Probability matrix for ant decisions
    
    // Ant colony
    Ant* d_ants;            // Device ant array
    int* d_best_tour;       // Best tour found
    float* d_best_length;   // Best tour length
    
    // Algorithm parameters
    int n_cities;           // Problem size
    int n_ants;             // Number of ants
    ACOAlgorithm algo_type;
    ProblemType prob_type;
    
    // CUDA random states
    hiprandState* d_rand_states;
    
    // Synchronization
    int* d_mutex;           // For atomic operations on shared best solution
};

// Kernel to initialize random states
__global__ void init_rand_states(hiprandState* states, unsigned long seed, int n_ants) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n_ants) {
        hiprand_init(seed, tid, 0, &states[tid]);
    }
}

// Kernel to initialize pheromone matrix
__global__ void init_pheromone(float* pheromone, int n, float init_val) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total = n * n;
    
    while (tid < total) {
        pheromone[tid] = init_val;
        tid += gridDim.x * blockDim.x;
    }
}

// Kernel to compute heuristic information (1/distance for TSP)
__global__ void compute_heuristic(float* heuristic, float* distance, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total = n * n;
    
    while (tid < total) {
        if (distance[tid] > 0) {
            heuristic[tid] = 1.0f / distance[tid];
        } else {
            heuristic[tid] = 0.0f;
        }
        tid += gridDim.x * blockDim.x;
    }
}

// Simplified greedy next city selection for debugging
__device__ int select_next_city_greedy(Ant* ant, float* distance, int n_cities) {
    int current = ant->current_city;
    float min_dist = FLT_MAX;
    int best_city = -1;
    
    for (int j = 0; j < n_cities; j++) {
        if (!ant->visited[j]) {
            float dist = distance[current * n_cities + j];
            if (dist < min_dist && dist > 0) {
                min_dist = dist;
                best_city = j;
            }
        }
    }
    
    return best_city;
}

// Device function to select next city using probability rule
__device__ int select_next_city(Ant* ant, float* pheromone, float* heuristic, 
                                hiprandState* rand_state, int n_cities, 
                                float alpha, float beta, float q0) {
    float q = hiprand_uniform(rand_state);
    int current = ant->current_city;
    
    // For debugging: Use greedy selection initially
    if (pheromone == nullptr) {
        return -1;
    }
    
    if (q < q0) {  // Exploitation (ACS)
        float max_val = -1.0f;
        int best_city = -1;
        
        for (int j = 0; j < n_cities; j++) {
            if (!ant->visited[j]) {
                float tau = pheromone[current * n_cities + j];
                float eta = heuristic[current * n_cities + j];
                
                // Check for valid values
                if (tau > 0 && eta > 0) {
                    float val = powf(tau, alpha) * powf(eta, beta);
                    if (val > max_val) {
                        max_val = val;
                        best_city = j;
                    }
                }
            }
        }
        return best_city;
    } else {  // Exploration
        // Calculate probabilities
        float sum = 0.0f;
        float probs[MAX_CITIES];
        
        for (int j = 0; j < n_cities; j++) {
            if (!ant->visited[j]) {
                float tau = pheromone[current * n_cities + j];
                float eta = heuristic[current * n_cities + j];
                
                if (tau > 0 && eta > 0) {
                    probs[j] = powf(tau, alpha) * powf(eta, beta);
                    sum += probs[j];
                } else {
                    probs[j] = 0.0f;
                }
            } else {
                probs[j] = 0.0f;
            }
        }
        
        if (sum == 0.0f) return -1;  // No valid city
        
        // Roulette wheel selection
        float r = hiprand_uniform(rand_state) * sum;
        float cumsum = 0.0f;
        
        for (int j = 0; j < n_cities; j++) {
            if (!ant->visited[j]) {
                cumsum += probs[j];
                if (cumsum >= r) {
                    return j;
                }
            }
        }
    }
    
    return -1;
}

// Kernel for ant solution construction
__global__ void construct_solutions(Ant* ants, float* pheromone, float* heuristic,
                                   float* distance, hiprandState* rand_states,
                                   int n_cities, int n_ants, float alpha, 
                                   float beta, float q0) {
    int ant_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (ant_id >= n_ants) return;
    
    Ant* ant = &ants[ant_id];
    hiprandState* rand_state = &rand_states[ant_id];
    
    // Initialize ant
    for (int i = 0; i < n_cities; i++) {
        ant->visited[i] = false;
        ant->tour[i] = -1;
    }
    ant->tour_size = 0;
    ant->tour_length = 0.0f;
    
    // Random starting city
    int start_city = hiprand(rand_state) % n_cities;
    ant->current_city = start_city;
    ant->tour[0] = start_city;
    ant->visited[start_city] = true;
    ant->tour_size = 1;
    
    // Construct tour
    for (int step = 1; step < n_cities; step++) {
        int next_city = select_next_city(ant, pheromone, heuristic, rand_state,
                                        n_cities, alpha, beta, q0);
        
        if (next_city >= 0 && next_city < n_cities) {
            // Add city to tour
            ant->tour[step] = next_city;
            ant->visited[next_city] = true;
            
            // Add distance
            int from = ant->current_city;
            int to = next_city;
            ant->tour_length += distance[from * n_cities + to];
            
            // Update current city
            ant->current_city = next_city;
            ant->tour_size++;
        } else {
            // If no valid city found, try to find any unvisited city
            for (int j = 0; j < n_cities; j++) {
                if (!ant->visited[j]) {
                    ant->tour[step] = j;
                    ant->visited[j] = true;
                    ant->tour_length += distance[ant->current_city * n_cities + j];
                    ant->current_city = j;
                    ant->tour_size++;
                    break;
                }
            }
        }
    }
    
    // Complete tour (return to start)
    if (ant->tour_size == n_cities && start_city >= 0 && start_city < n_cities) {
        ant->tour_length += distance[ant->current_city * n_cities + start_city];
    }
    
    // Debug: Check if tour is complete
    if (ant->tour_size != n_cities) {
        printf("Warning: Ant %d only visited %d cities\n", ant_id, ant->tour_size);
    }
}

// Kernel for local pheromone update (ACS)
__global__ void local_pheromone_update(float* pheromone, Ant* ants, int n_cities,
                                      int n_ants, float tau0, float phi) {
    int ant_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (ant_id >= n_ants) return;
    
    Ant* ant = &ants[ant_id];
    
    for (int i = 0; i < n_cities - 1; i++) {
        int from = ant->tour[i];
        int to = ant->tour[i + 1];
        if (from >= 0 && to >= 0) {
            int idx = from * n_cities + to;
            atomicAdd(&pheromone[idx], -phi * pheromone[idx] + phi * tau0);
        }
    }
}

// Kernel for global pheromone evaporation
__global__ void evaporate_pheromone(float* pheromone, int n, float rho) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total = n * n;
    
    while (tid < total) {
        pheromone[tid] *= (1.0f - rho);
        tid += gridDim.x * blockDim.x;
    }
}

// Kernel for depositing pheromone (best ant or rank-based)
__global__ void deposit_pheromone(float* pheromone, int* tour, float deposit,
                                 int n_cities) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < n_cities) {
        int from = tour[i];
        int to = tour[(i + 1) % n_cities];
        
        if (from >= 0 && to >= 0) {
            int idx = from * n_cities + to;
            atomicAdd(&pheromone[idx], deposit);
            // Symmetric for undirected graphs
            idx = to * n_cities + from;
            atomicAdd(&pheromone[idx], deposit);
        }
    }
}

// Kernel to find best ant
__global__ void find_best_ant(Ant* ants, int n_ants, int* best_ant_idx,
                             float* best_length) {
    __shared__ float shared_lengths[BLOCK_SIZE];
    __shared__ int shared_indices[BLOCK_SIZE];
    
    int tid = threadIdx.x;
    int ant_id = blockIdx.x * blockDim.x + tid;
    
    // Load ant lengths to shared memory
    if (ant_id < n_ants) {
        shared_lengths[tid] = ants[ant_id].tour_length;
        shared_indices[tid] = ant_id;
    } else {
        shared_lengths[tid] = FLT_MAX;
        shared_indices[tid] = -1;
    }
    __syncthreads();
    
    // Reduction to find minimum
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            if (shared_lengths[tid + stride] < shared_lengths[tid]) {
                shared_lengths[tid] = shared_lengths[tid + stride];
                shared_indices[tid] = shared_indices[tid + stride];
            }
        }
        __syncthreads();
    }
    
    // Write result
    if (tid == 0) {
        atomicMin((int*)best_length, __float_as_int(shared_lengths[0]));
        if (__int_as_float(atomicAdd((int*)best_length, 0)) == shared_lengths[0]) {
            *best_ant_idx = shared_indices[0];
        }
    }
}

// 2-opt local search kernel
__global__ void two_opt_ls(int* tour, float* distance, int n_cities,
                          float* tour_length, int max_iters) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    __shared__ bool improved;
    
    for (int iter = 0; iter < max_iters; iter++) {
        if (tid == 0) improved = false;
        __syncthreads();
        
        int i = tid;
        while (i < n_cities - 2) {
            for (int j = i + 2; j < n_cities; j++) {
                if ((j + 1) % n_cities == i) continue;
                
                int a = tour[i];
                int b = tour[i + 1];
                int c = tour[j];
                int d = tour[(j + 1) % n_cities];
                
                float current_dist = distance[a * n_cities + b] + 
                                   distance[c * n_cities + d];
                float new_dist = distance[a * n_cities + c] + 
                               distance[b * n_cities + d];
                
                if (new_dist < current_dist) {
                    // Reverse tour segment
                    int start = i + 1;
                    int end = j;
                    while (start < end) {
                        int temp = tour[start];
                        tour[start] = tour[end];
                        tour[end] = temp;
                        start++;
                        end--;
                    }
                    improved = true;
                    atomicAdd(tour_length, new_dist - current_dist);
                }
            }
            i += gridDim.x * blockDim.x;
        }
        
        __syncthreads();
        if (!improved) break;
    }
}

// Host functions for ACO management
// These are the implementations of the functions declared in cuda_aco.h

// Use extern "C" only when compiling as library for C++ compatibility
#ifdef ACO_LIBRARY_MODE
extern "C" {
#endif

ACOData* aco_init(int n_cities, int n_ants, ACOAlgorithm algo, ProblemType prob) {
    ACOData* aco = (ACOData*)malloc(sizeof(ACOData));
    aco->n_cities = n_cities;
    aco->n_ants = n_ants;
    aco->algo_type = algo;
    aco->prob_type = prob;
    
    printf("Initializing ACO: %d cities, %d ants\n", n_cities, n_ants);
    
    // Allocate device memory
    int matrix_size = n_cities * n_cities * sizeof(float);
    CUDA_CHECK(hipMalloc(&aco->d_distance, matrix_size));
    CUDA_CHECK(hipMalloc(&aco->d_pheromone, matrix_size));
    CUDA_CHECK(hipMalloc(&aco->d_heuristic, matrix_size));
    CUDA_CHECK(hipMalloc(&aco->d_prob, matrix_size));
    
    if (prob == QAP) {
        CUDA_CHECK(hipMalloc(&aco->d_flow, matrix_size));
    }
    
    // Allocate ant memory - need to properly manage device arrays
    Ant* h_ants = (Ant*)malloc(n_ants * sizeof(Ant));
    for (int i = 0; i < n_ants; i++) {
        CUDA_CHECK(hipMalloc(&h_ants[i].tour, n_cities * sizeof(int)));
        CUDA_CHECK(hipMalloc(&h_ants[i].visited, n_cities * sizeof(bool)));
        h_ants[i].tour_length = 0.0f;
        h_ants[i].current_city = 0;
        h_ants[i].tour_size = 0;
        
        // Initialize tour array to -1
        int* init_tour = (int*)malloc(n_cities * sizeof(int));
        for (int j = 0; j < n_cities; j++) {
            init_tour[j] = -1;
        }
        CUDA_CHECK(hipMemcpy(h_ants[i].tour, init_tour, n_cities * sizeof(int), 
                            hipMemcpyHostToDevice));
        free(init_tour);
    }
    CUDA_CHECK(hipMalloc(&aco->d_ants, n_ants * sizeof(Ant)));
    CUDA_CHECK(hipMemcpy(aco->d_ants, h_ants, n_ants * sizeof(Ant), 
                        hipMemcpyHostToDevice));
    free(h_ants);
    
    CUDA_CHECK(hipMalloc(&aco->d_best_tour, n_cities * sizeof(int)));
    CUDA_CHECK(hipMalloc(&aco->d_best_length, sizeof(float)));
    
    // Initialize random states
    CUDA_CHECK(hipMalloc(&aco->d_rand_states, n_ants * sizeof(hiprandState)));
    int blocks = (n_ants + BLOCK_SIZE - 1) / BLOCK_SIZE;
    init_rand_states<<<blocks, BLOCK_SIZE>>>(aco->d_rand_states, time(NULL), n_ants);
    
    // Initialize pheromone matrix
    blocks = (n_cities * n_cities + BLOCK_SIZE - 1) / BLOCK_SIZE;
    init_pheromone<<<blocks, BLOCK_SIZE>>>(aco->d_pheromone, n_cities, PHEROMONE_INIT);
    
    CUDA_CHECK(hipMalloc(&aco->d_mutex, sizeof(int)));
    CUDA_CHECK(hipMemset(aco->d_mutex, 0, sizeof(int)));
    
    return aco;
}

// Simple test kernel to verify basic functionality
__global__ void test_ant_init(Ant* ants, int n_cities, int n_ants) {
    int ant_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (ant_id >= n_ants) return;
    
    Ant* ant = &ants[ant_id];
    
    // Simple sequential tour for testing
    for (int i = 0; i < n_cities; i++) {
        ant->tour[i] = i;
        ant->visited[i] = true;
    }
    ant->tour_size = n_cities;
    ant->tour_length = 100.0f * ant_id;  // Dummy length
    ant->current_city = n_cities - 1;
}

void aco_load_problem(ACOData* aco, float* distance_matrix, float* flow_matrix) {
    int matrix_size = aco->n_cities * aco->n_cities * sizeof(float);
    
    printf("Loading problem data...\n");
    
    CUDA_CHECK(hipMemcpy(aco->d_distance, distance_matrix, matrix_size,
                        hipMemcpyHostToDevice));
    
    if (aco->prob_type == QAP && flow_matrix != NULL) {
        CUDA_CHECK(hipMemcpy(aco->d_flow, flow_matrix, matrix_size,
                            hipMemcpyHostToDevice));
    }
    
    // Compute heuristic information
    int blocks = (aco->n_cities * aco->n_cities + BLOCK_SIZE - 1) / BLOCK_SIZE;
    compute_heuristic<<<blocks, BLOCK_SIZE>>>(aco->d_heuristic, aco->d_distance,
                                             aco->n_cities);
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Verify distance matrix was loaded correctly
    float sample_dist;
    CUDA_CHECK(hipMemcpy(&sample_dist, aco->d_distance, sizeof(float),
                        hipMemcpyDeviceToHost));
    printf("Sample distance[0][0] = %.2f\n", sample_dist);
    
    // Test: Initialize ants with simple tours
    int ant_blocks = (aco->n_ants + BLOCK_SIZE - 1) / BLOCK_SIZE;
    test_ant_init<<<ant_blocks, BLOCK_SIZE>>>(aco->d_ants, aco->n_cities, aco->n_ants);
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Verify test initialization
    Ant h_test_ant;
    CUDA_CHECK(hipMemcpy(&h_test_ant, &aco->d_ants[0], sizeof(Ant),
                        hipMemcpyDeviceToHost));
    printf("Test ant tour size: %d, length: %.2f\n", 
           h_test_ant.tour_size, h_test_ant.tour_length);
}

void aco_run(ACOData* aco, int max_iterations, float* best_tour, float* best_length) {
    int ant_blocks = (aco->n_ants + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int matrix_blocks = (aco->n_cities * aco->n_cities + BLOCK_SIZE - 1) / BLOCK_SIZE;
    
    float h_best_length = FLT_MAX;
    int h_best_ant_idx = -1;
    
    // Initialize best tour to invalid values
    for (int i = 0; i < aco->n_cities; i++) {
        best_tour[i] = -1;
    }
    
    printf("Starting ACO with %d ants for %d cities\n", aco->n_ants, aco->n_cities);
    
    for (int iter = 0; iter < max_iterations; iter++) {
        // Construct solutions
        construct_solutions<<<ant_blocks, BLOCK_SIZE>>>(
            aco->d_ants, aco->d_pheromone, aco->d_heuristic, aco->d_distance,
            aco->d_rand_states, aco->n_cities, aco->n_ants, ALPHA, BETA, Q0
        );
        CUDA_CHECK(hipDeviceSynchronize());
        
        // Check for CUDA errors
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA error in construct_solutions: %s\n", hipGetErrorString(err));
            break;
        }
        
        // Local search (optional) - fixed to work with device memory
        if (iter % 10 == 0 && aco->n_ants > 0) {  // Apply every 10 iterations
            // Get all ant data from device
            Ant* h_ants_temp = (Ant*)malloc(aco->n_ants * sizeof(Ant));
            CUDA_CHECK(hipMemcpy(h_ants_temp, aco->d_ants, aco->n_ants * sizeof(Ant),
                                hipMemcpyDeviceToHost));
            
            // Apply local search to each ant's tour on device
            for (int a = 0; a < aco->n_ants; a++) {
                // Note: h_ants_temp[a].tour is a device pointer
                // Call local search kernel directly with device pointer
                two_opt_ls<<<1, 32>>>(h_ants_temp[a].tour, aco->d_distance, 
                                     aco->n_cities, &aco->d_ants[a].tour_length, 100);
            }
            CUDA_CHECK(hipDeviceSynchronize());
            free(h_ants_temp);
        }
        
        // Find best ant - fixed version
        float* d_best_ant_length = nullptr;
        int* d_best_ant_idx = nullptr;
        CUDA_CHECK(hipMalloc(&d_best_ant_length, sizeof(float)));
        CUDA_CHECK(hipMalloc(&d_best_ant_idx, sizeof(int)));
        
        float max_float = FLT_MAX;
        int neg_one = -1;
        CUDA_CHECK(hipMemcpy(d_best_ant_length, &max_float, sizeof(float),
                            hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_best_ant_idx, &neg_one, sizeof(int),
                            hipMemcpyHostToDevice));
        
        find_best_ant<<<ant_blocks, BLOCK_SIZE>>>(aco->d_ants, aco->n_ants,
                                                 d_best_ant_idx, d_best_ant_length);
        CUDA_CHECK(hipDeviceSynchronize());
        
        // Get best ant index and length
        CUDA_CHECK(hipMemcpy(&h_best_ant_idx, d_best_ant_idx, sizeof(int),
                            hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(&h_best_length, d_best_ant_length, sizeof(float),
                            hipMemcpyDeviceToHost));
        
        // Debug: Check ant tours
        if (iter == 0 || iter % 100 == 0) {
            printf("Iteration %d: ", iter);
            if (h_best_ant_idx >= 0) {
                printf("Best ant %d, length = %.2f\n", h_best_ant_idx, h_best_length);
            } else {
                printf("No valid ant found!\n");
                
                // Debug: Check first ant's tour
                Ant h_ant_debug;
                CUDA_CHECK(hipMemcpy(&h_ant_debug, &aco->d_ants[0], sizeof(Ant),
                                    hipMemcpyDeviceToHost));
                int* h_tour_debug = (int*)malloc(aco->n_cities * sizeof(int));
                CUDA_CHECK(hipMemcpy(h_tour_debug, h_ant_debug.tour,
                                    aco->n_cities * sizeof(int),
                                    hipMemcpyDeviceToHost));
                printf("First ant tour: ");
                for (int i = 0; i < std::min(10, aco->n_cities); i++) {
                    printf("%d ", h_tour_debug[i]);
                }
                printf("... (length: %.2f, size: %d)\n", 
                       h_ant_debug.tour_length, h_ant_debug.tour_size);
                free(h_tour_debug);: ");
                for (int i = 0; i < min(10, aco->n_cities); i++) {
                    printf("%d ", h_tour_debug[i]);
                }
                printf("... (length: %.2f, size: %d)\n", 
                       h_ant_debug.tour_length, h_ant_debug.tour_size);
                free(h_tour_debug);
            }
        }
        
        // Update pheromones
        evaporate_pheromone<<<matrix_blocks, BLOCK_SIZE>>>(aco->d_pheromone,
                                                          aco->n_cities, RHO);
        
        // Deposit pheromone based on algorithm type
        if (h_best_ant_idx >= 0 && h_best_ant_idx < aco->n_ants) {
            // Get the best ant's data
            Ant h_best_ant_meta;
            CUDA_CHECK(hipMemcpy(&h_best_ant_meta, &aco->d_ants[h_best_ant_idx],
                                sizeof(Ant), hipMemcpyDeviceToHost));
            
            float deposit = 1.0f / h_best_length;
            
            switch (aco->algo_type) {
                case AS:
                    // All ants deposit - use device pointers directly
                    for (int a = 0; a < aco->n_ants; a++) {
                        Ant h_ant_meta;
                        CUDA_CHECK(hipMemcpy(&h_ant_meta, &aco->d_ants[a], sizeof(Ant),
                                            hipMemcpyDeviceToHost));
                        deposit_pheromone<<<1, aco->n_cities>>>(aco->d_pheromone,
                                                               h_ant_meta.tour,  // This is device pointer
                                                               1.0f / h_ant_meta.tour_length,
                                                               aco->n_cities);
                    }
                    break;
                    
                case EAS:
                    // All ants + elite ants deposit more
                    for (int a = 0; a < aco->n_ants; a++) {
                        Ant h_ant_meta;
                        CUDA_CHECK(hipMemcpy(&h_ant_meta, &aco->d_ants[a], sizeof(Ant),
                                            hipMemcpyDeviceToHost));
                        deposit_pheromone<<<1, aco->n_cities>>>(aco->d_pheromone,
                                                               h_ant_meta.tour,
                                                               1.0f / h_ant_meta.tour_length,
                                                               aco->n_cities);
                    }
                    // Elite ant deposits extra
                    deposit_pheromone<<<1, aco->n_cities>>>(aco->d_pheromone,
                                                           h_best_ant_meta.tour,
                                                           deposit * 5.0f,  // Elite weight
                                                           aco->n_cities);
                    break;
                    
                case MMAS:
                    // Only best ant deposits, with min/max limits
                    deposit_pheromone<<<1, aco->n_cities>>>(aco->d_pheromone,
                                                           h_best_ant_meta.tour,
                                                           deposit, aco->n_cities);
                    break;
                    
                case ACS:
                    // Global best deposits
                    deposit_pheromone<<<1, aco->n_cities>>>(aco->d_pheromone,
                                                           h_best_ant_meta.tour,
                                                           deposit, aco->n_cities);
                    // Also apply local updates during construction
                    local_pheromone_update<<<ant_blocks, BLOCK_SIZE>>>(
                        aco->d_pheromone, aco->d_ants, aco->n_cities,
                        aco->n_ants, PHEROMONE_INIT, 0.1f
                    );
                    break;
                    
                default:
                    // Default to AS behavior
                    break;
            }
        }
        
        CUDA_CHECK(hipDeviceSynchronize());
        
        // Clean up temporary allocations
        CUDA_CHECK(hipFree(d_best_ant_length));
        CUDA_CHECK(hipFree(d_best_ant_idx));
        
        // Print iteration info
        if (iter % 10 == 0) {
            printf("Iteration %d: Best length = %.2f\n", iter, h_best_length);
        }
    }
    
    // Copy best solution to host - fixed version
    *best_length = h_best_length;
    if (h_best_ant_idx >= 0 && h_best_ant_idx < aco->n_ants) {
        Ant h_best_ant_meta;
        CUDA_CHECK(hipMemcpy(&h_best_ant_meta, &aco->d_ants[h_best_ant_idx],
                            sizeof(Ant), hipMemcpyDeviceToHost));
        
        // Copy the tour data from device to host
        int* h_tour_temp = (int*)malloc(aco->n_cities * sizeof(int));
        CUDA_CHECK(hipMemcpy(h_tour_temp, h_best_ant_meta.tour,
                            aco->n_cities * sizeof(int),
                            hipMemcpyDeviceToHost));
        
        // Convert to float array (as expected by interface)
        for (int i = 0; i < aco->n_cities; i++) {
            best_tour[i] = (float)h_tour_temp[i];
        }
        free(h_tour_temp);
    }
}

void aco_cleanup(ACOData* aco) {
    // Free ant memory - properly handle device pointers
    Ant* h_ants = (Ant*)malloc(aco->n_ants * sizeof(Ant));
    CUDA_CHECK(hipMemcpy(h_ants, aco->d_ants, aco->n_ants * sizeof(Ant),
                        hipMemcpyDeviceToHost));
    
    for (int i = 0; i < aco->n_ants; i++) {
        CUDA_CHECK(hipFree(h_ants[i].tour));
        CUDA_CHECK(hipFree(h_ants[i].visited));
    }
    free(h_ants);
    
    CUDA_CHECK(hipFree(aco->d_ants));
    CUDA_CHECK(hipFree(aco->d_distance));
    CUDA_CHECK(hipFree(aco->d_pheromone));
    CUDA_CHECK(hipFree(aco->d_heuristic));
    CUDA_CHECK(hipFree(aco->d_prob));
    
    if (aco->prob_type == QAP) {
        CUDA_CHECK(hipFree(aco->d_flow));
    }
    
    CUDA_CHECK(hipFree(aco->d_best_tour));
    CUDA_CHECK(hipFree(aco->d_best_length));
    CUDA_CHECK(hipFree(aco->d_rand_states));
    CUDA_CHECK(hipFree(aco->d_mutex));
    
    free(aco);
}

#ifdef ACO_LIBRARY_MODE
} // extern "C"
#endif

// Remove the old extern "C" declarations since they're now in the header
// The functions are already wrapped in extern "C" above

// Standalone test function (only compiled when in standalone mode)
#ifdef STANDALONE_BUILD

int main(int argc, char** argv) {
    // Example: small TSP instance
    int n_cities = 100;
    int n_ants = 128;
    int max_iterations = 1000;
    
    // Generate random distance matrix for testing
    float* distance_matrix = (float*)malloc(n_cities * n_cities * sizeof(float));
    srand(42);
    for (int i = 0; i < n_cities; i++) {
        for (int j = 0; j < n_cities; j++) {
            if (i == j) {
                distance_matrix[i * n_cities + j] = 0;
            } else if (i < j) {
                distance_matrix[i * n_cities + j] = (rand() % 100) + 1;
                distance_matrix[j * n_cities + i] = distance_matrix[i * n_cities + j];
            }
        }
    }
    
    // Initialize ACO
    printf("Initializing CUDA ACO for TSP with %d cities and %d ants\n",
           n_cities, n_ants);
    ACOData* aco = aco_init(n_cities, n_ants, ACS, TSP);
    
    // Load problem
    printf("Loading problem data...\n");
    aco_load_problem(aco, distance_matrix, NULL);
    
    // Run ACO
    float* best_tour = (float*)malloc(n_cities * sizeof(float));
    float best_length;
    
    printf("Running ACO for %d iterations...\n", max_iterations);
    clock_t start = clock();
    aco_run(aco, max_iterations, best_tour, &best_length);
    clock_t end = clock();
    
    double cpu_time = ((double)(end - start)) / CLOCKS_PER_SEC;
    printf("\nOptimization complete!\n");
    printf("Best tour length: %.2f\n", best_length);
    printf("Execution time: %.2f seconds\n", cpu_time);
    
    // Cleanup
    aco_cleanup(aco);
    free(distance_matrix);
    free(best_tour);
    
    return 0;
}

#endif // STANDALONE_BUILD